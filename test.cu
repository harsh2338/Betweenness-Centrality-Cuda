// %%cu
#include <bits/stdc++.h>
using namespace std;
class Graph {

public:

	int numNodes, numEdges;
	int *adjList, *adjListPointers;
	int *edgeList1, *edgeList2;

	int getNumNodes() {
		return numNodes;
	}

	int getNumEdges() {
		return numEdges;
	}

	void readGraph() {
		cin >> numNodes >> numEdges;
		adjListPointers = new int[numNodes +1];
		adjList = new int[2 * numEdges +1];
		for(int i=0; i<(2 * numEdges); i++)
			cin >> adjList[i];
		for(int i=0; i<=numNodes; i++) 
			cin >> adjListPointers[i];
	}

	void convertToCOO() {
		edgeList2 = adjList;
		edgeList1 = new int[2 * numEdges +1];

		for(int i=0; i <numNodes; ++i) {
			for(int j=adjListPointers[i]; j<adjListPointers[i+1]; ++j){
				edgeList1[j] = i;
			}
		}
	}
	int *getadjList(int node) {
		return adjList;
	}

	int *getadjListPointers(int node) {
		return adjListPointers;
	}

};
#include <iostream>
#include <hip/hip_runtime.h>
#define MAX_THREAD_COUNT 1024
#define MAX_MEMORY ((long long)4e9)

void make_graph(int LIM){
        int x, y;
    int n=LIM;
    int m=LIM;
    int cur = 0;

    freopen("input.txt", "w", stdout);
    cout << n << " " << m << endl;

    vector<int> *v = new vector<int>[n+1];
    for (int i = 0; i < m; i++) {
        do {
            x = rand() % n;
            y = rand() % n;
        } while (x == y);

        v[x].push_back(y);
        v[y].push_back(x);
    }

    for(int i=0; i<n; i++)
        for(int node : v[i])
            cout<<node<<" ";
    cout<<"\n";

    cout<<"0 ";
    for(int i=0; i<n; i++) {
        cur += v[i].size();
        cout<<cur<<" ";
    }
    cout<<"\n";
}

using namespace std;

__global__ void betweennessCentralityKernel(Graph *graph, float *bc, int nodeCount,
            int *sigma, int *distance, float *dependency, int *Q, int *Qpointers) {
    
    int idx = threadIdx.x;
    if(idx >= nodeCount)
        return;
    
    __shared__ int s;
    __shared__ int Q_len;
    __shared__ int Qpointers_len;
    __shared__ int noOfBlocks;

    if(idx == 0) {
        s = blockIdx.x - gridDim.x;
        noOfBlocks = gridDim.x;
    }
    __syncthreads();
    
    while(s < nodeCount - noOfBlocks)
    {
        if(idx == 0)
        {
            s += noOfBlocks;

            Q[0 + (blockIdx.x * nodeCount)] = s;
            Q_len = 1;
            Qpointers[0 + (blockIdx.x * nodeCount)] = 0;
            Qpointers[1 + (blockIdx.x * nodeCount)] = 1;
            Qpointers_len = 1;
        }
        __syncthreads();

        for(int v=idx; v<nodeCount; v+=blockDim.x)
        {
            if(v == s)
            {
                distance[v + (blockIdx.x * nodeCount)] = 0;
                sigma[v + (blockIdx.x * nodeCount)] = 1;
            }
            else
            {
                distance[v + (blockIdx.x * nodeCount)] = INT_MAX;
                sigma[v + (blockIdx.x * nodeCount)] = 0;
            }
            dependency[v + (blockIdx.x * nodeCount)] = 0.0;
        }
        __syncthreads();
        
        while(true)
        {
            __syncthreads();
            for(int k=idx; k<Qpointers[Qpointers_len + (blockIdx.x * nodeCount)]; k+=blockDim.x) 
            {
                if(k < Qpointers[Qpointers_len -1 + (blockIdx.x * nodeCount)])
                    continue;

                int v = Q[k + (blockIdx.x * nodeCount)];
                for(int r = graph->adjListPointers[v]; r < graph->adjListPointers[v + 1]; r++)
                {
                    int w = graph->adjList[r];
                    if(atomicCAS(&distance[w + (blockIdx.x * nodeCount)], INT_MAX, distance[v + (blockIdx.x * nodeCount)] +1) == INT_MAX)
                    {
                        int t = atomicAdd(&Q_len, 1);
                        Q[t + (blockIdx.x * nodeCount)] = w;
                    }
                    if(distance[w + (blockIdx.x * nodeCount)] == (distance[v + (blockIdx.x * nodeCount)]+1))
                    {
                        atomicAdd(&sigma[w + (blockIdx.x * nodeCount)], sigma[v + (blockIdx.x * nodeCount)]);
                    }
                }
            }
            __syncthreads();

            if(Q_len == Qpointers[Qpointers_len + (blockIdx.x * nodeCount)])
                break;

            if(idx == 0)
            {
                Qpointers_len++;
                Qpointers[Qpointers_len + (blockIdx.x * nodeCount)] = Q_len;
            }
            __syncthreads();
        }
        __syncthreads();
        
        while(Qpointers_len > 0)
        {
            for(int k=idx; k < Qpointers[Qpointers_len + (blockIdx.x * nodeCount)]; k+=blockDim.x) 
            {
                if(k < Qpointers[Qpointers_len -1 + (blockIdx.x * nodeCount)])
                    continue;

                int v = Q[k + (blockIdx.x * nodeCount)];
                for(int r = graph->adjListPointers[v]; r < graph->adjListPointers[v + 1]; r++)
                {
                    int w = graph->adjList[r];
                    if(distance[w + (blockIdx.x * nodeCount)] == (distance[v + (blockIdx.x * nodeCount)] + 1))
                    {
                        if (sigma[w + (blockIdx.x * nodeCount)] != 0)
                            dependency[v + (blockIdx.x * nodeCount)] += (sigma[v + (blockIdx.x * nodeCount)] * 1.0 / sigma[w + (blockIdx.x * nodeCount)]) * (1 + dependency[w + (blockIdx.x * nodeCount)]);
                    }
                }
                if (v != s)
                {
                    atomicAdd(bc + v, dependency[v + (blockIdx.x * nodeCount)] / 2);
                }
            }
            __syncthreads();

            if(idx == 0)
                Qpointers_len--;

            __syncthreads();
        }
    }
}

float *betweennessCentrality(Graph *graph, int nodeCount)
{
    float *bc = new float[nodeCount]();
    float *device_bc, *dependency;
    int *sigma, *distance, *Q, *Qpointers;

    const int BLOCK_COUNT = MAX_MEMORY / (4 * 5 * nodeCount);

    hipMalloc((void **)&device_bc, sizeof(float) * nodeCount);
    hipMalloc((void **)&sigma, sizeof(int) * nodeCount * BLOCK_COUNT);
    hipMalloc((void **)&distance, sizeof(int) * nodeCount * BLOCK_COUNT);
    hipMalloc((void **)&Q, sizeof(int) * (nodeCount) * BLOCK_COUNT);
    hipMalloc((void **)&Qpointers, sizeof(int) * (nodeCount) * BLOCK_COUNT);
    hipMalloc((void **)&dependency, sizeof(float) * nodeCount * BLOCK_COUNT);
    hipMemcpy(device_bc, bc, sizeof(float) * nodeCount, hipMemcpyHostToDevice);

    betweennessCentralityKernel<<<BLOCK_COUNT, MAX_THREAD_COUNT>>>(graph, device_bc, nodeCount, sigma, distance, dependency, Q, Qpointers);
    hipDeviceSynchronize();

    hipMemcpy(bc, device_bc, sizeof(float) * nodeCount, hipMemcpyDeviceToHost);
    hipFree(device_bc);
    hipFree(sigma);
    hipFree(dependency);
    hipFree(distance);
    hipFree(Q);
    hipFree(Qpointers);
    return bc;
}
double getWD(){
        freopen("input.txt", "r", stdin);

    Graph *graph = new Graph();
    Graph *cudaGraph;

    hipMalloc((void **)&cudaGraph, sizeof(Graph));
    graph->readGraph();

    int numNodes = graph->getNumNodes();
    int numEdges = graph->getNumEdges();
    hipMemcpy(cudaGraph, graph, sizeof(Graph), hipMemcpyHostToDevice);

    int *adjList;
    hipMalloc((void **)&adjList, sizeof(int) * (2 * numEdges + 1));
    hipMemcpy(adjList, graph->adjList, sizeof(int) * (2 * numEdges + 1), hipMemcpyHostToDevice);
    hipMemcpy(&(cudaGraph->adjList), &adjList, sizeof(int *), hipMemcpyHostToDevice);

    int *adjListPointers;
    hipMalloc((void **)&adjListPointers, sizeof(int) * (numNodes + 1));
    hipMemcpy(adjListPointers, graph->adjListPointers, sizeof(int) * (numNodes + 1), hipMemcpyHostToDevice);
    hipMemcpy(&(cudaGraph->adjListPointers), &adjListPointers, sizeof(int *), hipMemcpyHostToDevice);
 
    clock_t start, end;
    start = clock();
    float *bc = betweennessCentrality(cudaGraph, numNodes);
    end = clock();
    double time_taken = 1000.0 * (end - start) / (double)CLOCKS_PER_SEC;
    double maxBetweenness = -1, maxBetweennessNode=-1;
    cout<<"Time taken :"<<time_taken<<"\n";
    // cout<<"Betweenness Centrality for each node:\n";
    for (int i = 0; i < numNodes; i++)
    {
        if(maxBetweenness<bc[i]){
            maxBetweenness = bc[i];
            maxBetweennessNode=i;
        }
        // cout<<i<<" : "<<bc[i]<<"\n";
    }
    cout<<"Maximum betweenness centrality : ";
    cout<<maxBetweenness<<"\n";
    cout<<"Node with Maximum betweenness centrality : ";
    cout<<maxBetweennessNode<<"\n";

    hipFree(adjList);
    hipFree(adjListPointers);
    hipFree(cudaGraph);
    return time_taken;
}
int main(){
    vector<double>x,v,e,we,wd;
    for(int i=500;i<=1500;i+=500){
        make_graph(i);
        x.push_back(i);
        //v.push_back(getVertex());
        //e.push_back(getEdge());
        //we.push_back(getWE());
        wd.push_back(getWD());
    }
    for(int i:x)cout<<i<<" , ";
    cout<<"\n\n";
    for(int i:v)cout<<i<<" , ";
        cout<<"\n\n";


    for(int i:e)cout<<i<<" , ";

    cout<<"\n\n";

    for(int i:we)cout<<i<<" , ";
    cout<<"\n\n";

    for(int i:wd)cout<<i<<" , ";
    cout<<"\n\n";
    
    
}